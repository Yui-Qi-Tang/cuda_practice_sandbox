#include "cuda_op.h"

// Constructor
CudaChecker::CudaChecker(const int devices) {
	_usedDevices = devices;
	_deviceCounts = deviceCounts();

	if(_usedDevices > _deviceCounts) {
        printf("These are not %d devices(Max devices: %d)", _usedDevices, _deviceCounts);
	}
	getDeviceProps();
    // printf("Initial CUDA ")
	
}

// Destructor
CudaChecker::~CudaChecker() {
	// This is blank;
}

// initilize CUDA and get device count
int CudaChecker::initCuda(const int max_device_count_to_use){
	int deviceCount=0;
	int cuda_err_code = -1; // not set
	cuda_err_code = hipGetDeviceCount(&deviceCount);
	if(deviceCount==0){
		fprintf(stderr, "There is no device. and cuda_code is: %d\n", cuda_err_code);
		return 0;
	}
	else {
		fprintf(stdout, "device count:%d\n", deviceCount);
	}
	
	if(deviceCount>max_device_count_to_use){
		fprintf(
			stdout,
			"deviceCount (%d) > max_device_count_to_use (%d), use first some GPUs only.\n",
			deviceCount, max_device_count_to_use
		);
		deviceCount = max_device_count_to_use;
	}
	
	// Check if EVERY devices support CUDA
	bool all_device_sup = true;
	hipDeviceProp_t prop;
	for(int i = 0; i < deviceCount; i++) {
		hipError_t val = hipGetDeviceProperties(&prop, i);
		if(val != hipSuccess || prop.major<2){
			all_device_sup = false;
			break;
		}
	}
	
	if(all_device_sup==true){
		fprintf(stdout, "CUDA prop.major:%d\n", prop.major);
	}
	else {
		fprintf(stderr, "Not all devices supporting CUDA.\n");
		return 0;
	}
	
	hipSetDevice(0);
    printf("hipSetDevice %d/%d\n", 0, deviceCount);
	return deviceCount;
}


/*Get device counts on your platform*/
int CudaChecker::deviceCounts() {
	int deviceCounts = 0;
	int cudaErrCode = -1; // not set
	cudaErrCode = hipGetDeviceCount(&deviceCounts);
	if (cudaErrCode != hipSuccess) {
		fprintf(stderr, "There is no device. and cuda error code is: %d\n", cudaErrCode);
        exit(-1);
	}	
	return deviceCounts;
}

int CudaChecker::getMaxDeviceCounts() {
	return _deviceCounts;
}

void CudaChecker::getDeviceProps() {
	_devProp = (hipDeviceProp_t *)malloc(sizeof(hipDeviceProp_t) * _usedDevices);
	// cuda device 由 0 開始計算，第一個裝置在0的位置
	for(int i = 0; i < _usedDevices; i++) {
        hipError_t status = hipGetDeviceProperties((_devProp+i), i);
		if(status != hipSuccess) {
            printf("Fatal Error: Get device properites failed!\n");
			exit(-1);
		} // fi
	} // for
}

void CudaChecker::dumpDevicesProperty() {
	/*
		*Use foreach??
		All fields of device property:
	      char name[256];
          size_t totalGlobalMem;
          size_t sharedMemPerBlock;
          size_t memPitch;
          size_t totalConstMem;
		  size_t textureAlignment;
		  

          int regsPerBlock;
          int warpSize;
          int maxThreadsPerBlock;
          int maxThreadsDim[3];
          int maxGridSize[3];
          int major;
          int minor;
          int clockRate;
          int deviceOverlap;
          int multiProcessorCount;
          int kernelExecTimeoutEnabled;
          int integrated;
          int canMapHostMemory;
          int computeMode;
          int concurrentKernels;
          int ECCEnabled;
          int pciBusID;
          int pciDeviceID;
          int tccDriver;
	*/
	hipDeviceProp_t *devPtr;
	for(int i = 0; i < _usedDevices; i++) {
		devPtr = _devProp + i;
		printf("==>Property of Device no.%d\n", i);
		printf("major: %d\n", devPtr -> major);
		printf("name: %s\n", devPtr -> name);
		printf("regsPerBlock: %d\n", devPtr -> regsPerBlock);
		printf("warpSize: %d\n", devPtr -> warpSize);
		printf("maxThreadsPerBlock: %d\n", devPtr -> maxThreadsPerBlock);
		printf("minor: %d\n", devPtr -> minor);
		printf("clockRate: %d\n", devPtr -> clockRate);
		printf("deviceOverlap: %d\n", devPtr -> deviceOverlap);
		printf("multiProcessorCount: %d\n", devPtr -> multiProcessorCount);
		printf("kernelExecTimeoutEnabled: %d\n", devPtr -> kernelExecTimeoutEnabled);
		printf("integrated: %d\n", devPtr -> integrated);
		printf("canMapHostMemory: %d\n", devPtr -> canMapHostMemory);
		printf("computeMode: %d\n", devPtr -> computeMode);
		printf("concurrentKernels: %d\n", devPtr -> concurrentKernels);
		printf("ECCEnabled: %d\n", devPtr -> ECCEnabled);
		printf("pciBusID: %d\n", devPtr -> pciBusID);
		printf("pciDeviceID: %d\n", devPtr -> pciDeviceID);
		printf("tccDriver: %d\n", devPtr -> tccDriver);
		
		for (int i = 0; i<3; i++) {
		    printf("maxThreadsDim[%d]: %d\n", i, devPtr -> maxThreadsDim[i] );
		}

		for (int i = 0; i<3; i++) {
		    printf("maxGridSize[%d]: %d\n", i, devPtr -> maxGridSize[i]);
		}
		
		printf("totalGlobalMem: %lu\n", devPtr -> totalGlobalMem);
        printf("sharedMemPerBlock: %lu\n", devPtr -> sharedMemPerBlock);
        printf("memPitch: %lu\n", devPtr -> memPitch);
        printf("totalConstMem: %lu\n", devPtr -> totalConstMem);
		printf("textureAlignment: %lu\n", devPtr -> textureAlignment);
	}
}
